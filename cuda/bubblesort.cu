#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include "win-gettimeofday.h"

#define THREADS_PER_BLOCK 512

__global__ void bubbleSort(int array[], int End) 
{ 
    int swapped = 0;
    int temp;
    do 
    {
        swapped = 0;
        for (int i = 0; i < End; i++) 
        {
            if (array[i] > array[i + 1]) 
            {
                temp = array[i];
                array[i] = array[i + 1];
                array[i + 1] = temp;
                swapped = 1;
            }
        }

    } while (swapped == 1);

}

void populateRandomArray(int *x, int num_elements) 
{
    for (int i = 0; i < num_elements; i++) 
    {
        x[i] = rand() % 100 + 1;
    }
}

void bubbleSortCPU(int array[], int End) 
{ 
    int swapped = 0;
    int temp;

    do 
    {
        swapped = 0;
        for (int i = 0; i < End; i++) 
        {
            if (array[i] > array[i + 1]) 
            {
                temp = array[i];
                array[i] = array[i + 1];
                array[i + 1] = temp;
                swapped = 1;
            }
        }

    } while (swapped == 1);

}

int main(void) 
{
    const int number_of_elements = 100000; 

    int trials[number_of_elements]; 

    int *host_a;
    int *host_c;

    int *device_a;
    int *device_c;

    double cpu_time_without_allocation;
    double cpu_time_with_allocation;
    double cpu_end_time;

    double gpu_time_without_transfer;
    double gpu_time_with_transfer;
    double gpu_end_time_without_transfer;
    double gpu_end_time_with_transfer;

    for (int i = 0; i < 1; i++) 
    {
        int size = trials[i] *
                   sizeof(int); 

        int end = number_of_elements;

        host_a = (int *) malloc(size);
        host_c = (int *) malloc(size);

        hipMalloc((void **) &device_a, size);
        hipMalloc((void **) &device_c, size);

        populateRandomArray(host_a, number_of_elements);

        gpu_time_with_transfer = get_current_time(); 

        hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);

        gpu_time_without_transfer = get_current_time(); 

        dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
        dim3 dimGrid((trials[i] + dimBlock.x - 1) / dimBlock.x, 1, 1);

        bubbleSort << < dimGrid, dimBlock >> > (device_a, end); 

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) 
        {
            printf("Error: %s\n", hipGetErrorString(error));
        }

        hipDeviceSynchronize(); 

        gpu_end_time_without_transfer = get_current_time(); 

        hipMemcpy(host_c, device_a, size, hipMemcpyDeviceToHost);

        gpu_end_time_with_transfer = get_current_time();  

        printf("Number of elements = %d, GPU Time (Not including data transfer): %lfs\n", number_of_elements,
               (gpu_end_time_without_transfer - gpu_time_without_transfer));
        printf("Number of elements = %d, GPU Time (Including data transfer): %lfs\n", number_of_elements,
               (gpu_end_time_with_transfer - gpu_time_with_transfer));

        free(host_a);
        free(host_c);

        hipFree(device_a);
        hipFree(device_c);

        hipDeviceReset();
    }
    return 0;
}